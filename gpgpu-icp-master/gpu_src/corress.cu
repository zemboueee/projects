#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>

#include "icp.hh"

#define MAX_FLOAT 3.40282e+038

__global__ void search_corres(const float *p, const float *m, float *y,
                              size_t s)
{
    int i = (blockDim.x * blockIdx.x + threadIdx.x) * 3;
    if (i >= s)
        return;
    float pi[3] = { p[i], p[i + 1], p[i + 2] };

    float minD = MAX_FLOAT;
    size_t idx = 0;

    for (size_t k = 0; k < s; k += 3) {
        float mk[3] = { m[k], m[k + 1], m[k + 2] };

        float dist = (sqrt(pow(pi[0] - mk[0], 2) + pow(pi[1] - mk[1], 2) +
                           pow(pi[2] - mk[2], 2)));

        if (dist < minD) {
            minD = dist;
            idx = k;
        }
    }
    y[i] = m[idx];
    y[i + 1] = m[idx + 1];
    y[i + 2] = m[idx + 2];
}

Points get_correspondences(const Points p, const Points m)
{
    size_t size_malloc = p.size() * sizeof(float) * 3;

    float *cm, *cp, *cy, *arr_y, *arr_m, *arr_p;

    arr_p = p.convert_to_f();
    arr_m = m.convert_to_f();
    arr_y = (float *)std::malloc(size_malloc);

    hipMalloc((void **)&cp, size_malloc);
    hipMalloc((void **)&cm, size_malloc);
    hipMalloc((void **)&cy, size_malloc);

    hipMemcpy(cp, arr_p, size_malloc, hipMemcpyHostToDevice);
    hipMemcpy(cm, arr_m, size_malloc, hipMemcpyHostToDevice);

    search_corres<<<ceil(p.size() / 1024), 1024>>>(cp, cm, cy, p.size() * 3);
    hipDeviceSynchronize();

    hipMemcpy(arr_y, cy, size_malloc, hipMemcpyDeviceToHost);

    Points y(arr_y, p.size());

    free(arr_p);
    free(arr_m);
    free(arr_y);
    hipFree(cp);
    hipFree(cm);
    hipFree(cy);

    return y;
}
