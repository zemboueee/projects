#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>

#include "icp.hh"
#include "log.hh"
#include "points.hh"

int main(int argc, char *argv[])
{
    (void)argc;
    (void)argv;

    init_logs();
    Log::display = true;

    Log l("GPGPU ICP");
    l.title();

    Points p;
    Points model;

    if (argc < 2) {
        p = Points("../data/data_students/cow_ref.txt");
        model = Points("../data/data_students/cow_tr2.txt");
    } else {
        p = Points(argv[1]);
        model = Points(argv[2]);
    }

    l << "Parse OK" << std::endl;

    if (p.size() != model.size()) {
        Log l("Parsing", true);
        l << "Points sets need to have the same number of points" << std::endl;
        return 1;
    }

    apply_alignment(p, model);

    l << "Ending..." << std::endl;

    return 0;
}
