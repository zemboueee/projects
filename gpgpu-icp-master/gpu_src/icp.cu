#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <fstream>
#include <iostream>

#include "icp.hh"
#include "log.hh"

#define MAX_ITER 15
#define THRESH 0.00001

std::clock_t start_timer()
{
    return std::clock();
}

double stop_timer(std::clock_t start)
{
    return (std::clock() - start) / (double)CLOCKS_PER_SEC;
}

//s, R, t, err
alignment_t find_alignment(Points p, Points y)
{
    Log l("Find Alignment");
    alignment_t alignment;

    Vect3f mu_p = get_mean(p);
    l << "mu p: " << mu_p << std::endl;
    Vect3f mu_y = get_mean(y);
    l << "mu y: " << mu_y << std::endl;

    Points p_prime = create_prime(p, mu_p);
    Points y_prime = create_prime(y, mu_y);
    l << "primes ok" << std::endl;

    Matrix quaternion = get_quaternion_matrix(p_prime, y_prime);
    l << "quaternion: " << quaternion << std::endl;

    float scale = get_scaling_factor(p_prime, y_prime);
    l << "Scale: " << scale << std::endl;
    Matrix rotation = get_rotation_matrix(quaternion);
    l << "Rotation: " << rotation << std::endl;

    Matrix scaled_rotation(3);
    for (size_t i = 0; i < 3; i++)
        for (size_t j = 0; j < 3; j++)
            scaled_rotation[i][j] = rotation[i][j] * scale;

    l << "Rotation * scale: " << scaled_rotation << std::endl;

    Vect3f translation = get_transational_offset(mu_p, mu_y, scaled_rotation);
    l << "Translation: " << translation << std::endl;
    float error = residual_error(p, y, scaled_rotation, translation);
    l << "Residual error: " << error << std::endl;

    alignment.push_back(scale);
    alignment.push_back(rotation);
    alignment.push_back(scaled_rotation);
    alignment.push_back(translation);
    alignment.push_back(error);

    return alignment;
}

//s; R; t
Points apply_alignment(Points p, const Points model)
{
    size_t size = p.size();

    Log l("Alignment");
    Log lt("Timer");

    float final_scale = 1;
    Matrix final_rotation(3);
    final_rotation[0][0] = 1;
    final_rotation[1][1] = 1;
    final_rotation[2][2] = 1;
    Vect3f final_translation(0, 0, 0);
    float final_err = 0;

    for (size_t iter = 0; iter < MAX_ITER; iter++) {
        l.title(iter + 1);

        auto clk = start_timer();
        // Compute Y
        Points y = get_correspondences(p, model);

        lt << "Time Elapsed after get_correspondences(): " << stop_timer(clk)
           << "s\n";

        clk = start_timer();

        // Find Alignment
        alignment_t alignment = find_alignment(p, y);

        lt << "Time Elapsed after find_aligment(): " << stop_timer(clk)
           << "s\n";

        float scale = std::get<float>(alignment[0]);
        Matrix rotation = std::get<Matrix>(alignment[1]);
        Matrix scalled_rotation = std::get<Matrix>(alignment[2]);
        Vect3f translation = std::get<Vect3f>(alignment[3]);
        float err = std::get<float>(alignment[4]);

        // Saving final results
        final_scale *= scale;
        final_translation = final_translation + translation;
        final_rotation = final_rotation * rotation;

        // Applying (newP)
        Points newP = (scalled_rotation * p) + translation;

        // Error
        Vect3f e;

        for (size_t i = 0; i < size; i++) {
            e = y[i] - newP[i];
            err += e.x * e.x + e.y * e.y + e.z * e.z;
        }
        err /= size;
        l << "Error: " << err << std::endl;

        // Saving error
        final_err = err;

        if (final_err < THRESH) {
            break;
        }

        dump_on_file(newP, "result" + std::to_string(iter) + ".txt");

        // Applying to p
        p = newP;
    }
    Log lr("Result");
    lr.title();
    lr << "Final scale: " << final_scale << std::endl;
    lr << "Final rotation: " << final_rotation << std::endl;
    lr << "Final translation: " << final_translation << std::endl;
    lr << "Final error: " << final_err << std::endl;

    dump_on_file(p, "result.txt");

    return p;
}

void dump_on_file(Points p, std::string path)
{
    Log l("Dump on File");
    l << "Dump result on " << path << " | p_size:" << p.size() << std::endl;

    std::ofstream stream;
    stream.open(path);
    stream << "Points_0,Points_1,Points_2\n";

    for (size_t i = 0; i < p.size(); i++)
        stream << p[i].x << "," << p[i].y << "," << p[i].z << "\n";

    stream.close();
}
