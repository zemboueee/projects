#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>

#include "../Eigen/Core"
#include "../Eigen/Eigenvalues"
#include "../Eigen/SVD"
#include "icp.hh"
#include "log.hh"

Vect3f get_mean(Points points)
{
    Vect3f m(0, 0, 0);
    size_t s = points.size();
    for (size_t i = 0; i < s; i++) {
        m = m + points[i];
    }
    m.x /= s;
    m.y /= s;
    m.z /= s;

    return m;
}

Points create_prime(Points p, Vect3f mu)
{
    Points r = Points();
    for (size_t i = 0; i < p.size(); i++) {
        Vect3f tmp(p[i].x - mu.x, p[i].y - mu.y, p[i].z - mu.z);
        r.addPoint(tmp);
    }
    return r;
}

Matrix get_rotation_matrix(Matrix q)
{
    float q0 = q[0][0];
    float q1 = q[0][1];
    float q2 = q[1][0];
    float q3 = q[1][1];

    std::vector<float> Qb0 = { q0, q1, q2, q3 };
    std::vector<float> Qb1 = { -q1, q0, -q3, q2 };
    std::vector<float> Qb2 = { -q2, q3, q0, -q1 };
    std::vector<float> Qb3 = { -q3, -q2, q1, q0 };

    std::vector<float> Q0 = { q0, -q1, -q2, -q3 };
    std::vector<float> Q1 = { q1, q0, -q3, q2 };
    std::vector<float> Q2 = { q2, q3, q0, -q1 };
    std::vector<float> Q3 = { q3, -q2, q1, q0 };

    matrix_t QB_tmp = { Qb0, Qb1, Qb2, Qb3 };
    matrix_t Q_tmp = { Q0, Q1, Q2, Q3 };

    Matrix Q(Q_tmp);
    Matrix QB(QB_tmp);
    Matrix r(4, 4);

    for (size_t i = 0; i < 4; i++)
        for (size_t j = 0; j < 4; j++) {
            r[i][j] = 0;
            for (size_t k = 0; k < 4; k++)
                r[i][j] += QB[i][k] * Q[k][j];
        }

    Matrix r3(3, 3);

    for (size_t i = 0; i < 3; i++)
        for (size_t j = 0; j < 3; j++)
            r3[i][j] = r[1 + i][1 + j];

    return r3;
}

float getSum(Points Pprime, size_t P_idx, Points Yprime, size_t Y_idx)
{
    float sum = 0;
    for (size_t i = 0; i < Pprime.size(); i++)
        sum += Pprime[i][P_idx] * Yprime[i][Y_idx];

    return sum;
}

Matrix get_quaternion_matrix(Points Pprime, Points Yprime)
{
    Log l("Quaternion");

    float s_xx = getSum(Pprime, 0, Yprime, 0);
    float s_xy = getSum(Pprime, 0, Yprime, 1);
    float s_xz = getSum(Pprime, 0, Yprime, 2);

    float s_yx = getSum(Pprime, 1, Yprime, 0);
    float s_yy = getSum(Pprime, 1, Yprime, 1);
    float s_yz = getSum(Pprime, 1, Yprime, 2);

    float s_zx = getSum(Pprime, 2, Yprime, 0);
    float s_zy = getSum(Pprime, 2, Yprime, 1);
    float s_zz = getSum(Pprime, 2, Yprime, 2);

    Matrix N(4);
    N[0][0] = s_xx + s_yy + s_zz;
    N[0][1] = s_yz - s_zy;
    N[0][2] = s_zx - s_xz;
    N[0][3] = s_xy - s_yx;

    N[1][0] = s_yz - s_zy;
    N[1][1] = s_xx - s_zz - s_yy;
    N[1][2] = s_xy + s_yx;
    N[1][3] = s_xz + s_zx;

    N[2][0] = s_zx - s_xz;
    N[2][1] = s_yx + s_xy;
    N[2][2] = s_yy - s_zz - s_xx;
    N[2][3] = s_yz + s_zy;

    N[3][0] = s_xy - s_yx;
    N[3][1] = s_zx + s_xz;
    N[3][2] = s_zy + s_yz;
    N[3][3] = s_zz - s_yy - s_xx;

    l << "N: " << N << std::endl;

    double M[16] = { N[0][0], N[0][1], N[0][2], N[0][3], N[1][0], N[1][1],
                     N[1][2], N[1][3], N[2][0], N[2][1], N[2][2], N[2][3],
                     N[3][0], N[3][1], N[3][2], N[3][3] };
    Eigen::Matrix4d A_(M);
    Eigen::EigenSolver<Eigen::Matrix4d> es(A_);
    Eigen::MatrixXcd V_eigen = es.eigenvectors();

    //l << "V_eigen: " << std::endl << V_eigen << std::endl;

    Matrix R(2);
    R[0][0] = V_eigen.col(0)[0].real();
    R[0][1] = V_eigen.col(0)[1].real();
    R[1][0] = V_eigen.col(0)[2].real();
    R[1][1] = V_eigen.col(0)[3].real();

    return R;
}

float get_scaling_factor(Points Pprime, Points Yprime)
{
    float sp = 0;
    float d = 0;
    for (size_t i = 0; i < Yprime.size(); i++) {
        d += Yprime[i].x * Yprime[i].x + Yprime[i].y * Yprime[i].y +
             Yprime[i].z * Yprime[i].z;
        sp += Pprime[i].x * Pprime[i].x + Pprime[i].y * Pprime[i].y +
              Pprime[i].z * Pprime[i].z;
    }
    return std::sqrt(d / sp);
}

Vect3f get_transational_offset(Vect3f mu_p, Vect3f mu_y, Matrix R)
{
    Vect3f t(mu_y);
    for (size_t i = 0; i < 3; i++)
        for (size_t j = 0; j < 3; j++)
            t[i] -= R[i][j] * mu_p[j];

    return t;
}

float residual_error(const Points p, const Points y, Matrix sr, Vect3f t)
{
    float err = 0;

    Points newP = (sr * p) + t;

    for (size_t i = 0; i < p.size(); i++) {
        Vect3f d;
        d = y[i] - newP[i];
        err += d.x * d.x + d.y * d.y + d.z * d.z;
    }

    return err;
}
