#include "hip/hip_runtime.h"
#include <algorithm>
#include <fstream>
#include <iterator>
#include <limits>
#include <sstream>
#include <string>
#include <vector>

#include "log.hh"
#include "points.hh"

/*
** Points
*/

Points::Points(std::string path)
{
    Log l("Parsing");
    l << "Loading " << path << std::endl;

    std::ifstream file(path);

    std::string line = "";

    // Skip first line
    getline(file, line, '\n');

    while (getline(file, line, '\n')) {
        std::vector<std::string> vect;
        std::stringstream line_stream(line);
        while (line_stream.good()) {
            std::string value;
            getline(line_stream, value, ',');
            vect.push_back(value);
        }

        Vect3f v(std::stof(vect[0]), std::stof(vect[1]), std::stof(vect[2]));
        this->addPoint(v);
    }

    file.close();
}

Points::Points()
{
}

void Points::addPoint(Vect3f v)
{
    this->points_.push_back(v);
}

size_t Points::size() const
{
    return this->points_.size();
}

Vect3f Points::operator[](size_t i) const
{
    return this->points_[i];
}

Vect3f &Points::operator[](size_t i)
{
    return this->points_[i];
}

Points operator*(const Matrix &matrix, const Points &p)
{
    Points newP;
    for (size_t i = 0; i < p.size(); i++) {
        float x = p[i].x;
        float y = p[i].y;
        float z = p[i].z;
        newP.addPoint(
            Vect3f(x * matrix[0][0] + y * matrix[0][1] + z * matrix[0][2],
                   x * matrix[1][0] + y * matrix[1][1] + z * matrix[1][2],
                   x * matrix[2][0] + y * matrix[2][1] + z * matrix[2][2]));
    }
    return newP;
}

Points operator+(const Points &p, const Vect3f translation)
{
    Points newP;
    size_t s_size = p.size();

    for (size_t i = 0; i < s_size; i++) {
        newP.addPoint(p[i] + translation);
    }

    return newP;
}

/*
** Vect3f
*/

Vect3f::Vect3f(float x, float y, float z)
    : x{ x }
    , y{ y }
    , z{ z }
{
}

Vect3f::Vect3f()
    : Vect3f(0, 0, 0)
{
}

float &Vect3f::operator[](size_t i)
{
    switch (i % 3) {
    case 0:
        return x;
    case 1:
        return y;
    default:
        return z;
    }
}

Vect3f operator+(const Vect3f &v1, const Vect3f &v2)
{
    return Vect3f(v1.x + v2.x, v1.y + v2.y, v1.z + v2.z);
}

Vect3f operator-(const Vect3f &v1, const Vect3f &v2)
{
    return Vect3f(v1.x - v2.x, v1.y - v2.y, v1.z - v2.z);
}

/*
** Dump
*/

std::ostream &operator<<(std::ostream &os, const Vect3f &v)
{
    return os << "(" << v.x << ", " << v.y << ", " << v.z << ")";
}

std::ostream &operator<<(std::ostream &os, const Points &p)
{
    for (size_t i = 0; i < p.size(); i++) {
        os << std::endl << p[i];
    }
    return os;
}

float *Points::convert_to_f() const
{
    size_t size = this->size();
    size_t array_size = 3 * size;

    float *f = (float *)std::malloc(sizeof(float) * array_size);

    for (size_t i = 0; i < size; i++) {
        size_t j = 3 * i;
        f[j] = this->points_[i].x;
        f[j + 1] = this->points_[i].y;
        f[j + 2] = this->points_[i].z;
    }
    return f;
}

Points::Points(float *f, size_t s)
{
    Vect3f v(0, 0, 0);

    for (size_t i = 0; i < s * 3; i += 3) {
        v.x = f[i];
        v.y = f[i + 1];
        v.z = f[i + 2];

        this->addPoint(v);
    }
}
